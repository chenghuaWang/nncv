#include "hip/hip_runtime.h"
#include "libnncv/kernel/sgemm.cuh"

namespace nncv {
namespace rt {
namespace kernel {

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

__global__ void sgemm(float* __restrict__ lhs, float* __restrict__ rhs, float* __restrict__ dst,
                      int m, int n, int k) {
  // get all index needed
  int BX = blockIdx.x;
  int BY = blockIdx.y;
  int TX = threadIdx.x;
  int TY = threadIdx.y;

  // caculate all block based sizes
  const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
  const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
  const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;
  const int THIS_THREAD_ID = TY * THREAD_X_PER_BLOCK + TX;

  // how many sizes needed for register. Those for transfer global mem to shared mem.
  // globael mem -> register -> shared mem
  const int lhsTransfer = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_NUM_PER_BLOCK * 4);
  const int rhsTransfer = BLOCK_SIZE_K * BLOCK_SIZE_N / (THREAD_NUM_PER_BLOCK * 4);

  // create transfer register
  float lhsTransferRegister[4 * lhsTransfer];
  float rhsTransferRegister[4 * rhsTransfer];

  // [shared memory]. Double Buffer
  // contained in one block, shared by 16x16=256 threads in one block.
  __shared__ float lhsShared[2][BLOCK_SIZE_K][BLOCK_SIZE_M];  // 8 x 128 by default
  __shared__ float rhsShared[2][BLOCK_SIZE_K][BLOCK_SIZE_N];  // 8 x 128 by default

  // create the accumulate array on [register level]. And init it with 0.
  float accum[THREAD_SIZE_Y][THREAD_SIZE_X];  // 8 x 8 by default
#pragma unroll
  for (int i = 0; i < THREAD_SIZE_Y; i++) {
#pragma unroll
    for (int j = 0; j < THREAD_SIZE_X; j++) { accum[i][j] = 0.f; }
  }

  // create the register array for lhs and rhs. [register level] with double buffer
  float lhsRegister[2][THREAD_SIZE_Y];
  float rhsRegister[2][THREAD_SIZE_X];

  // prepare for transfer
  const int LHS_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
  const int RHS_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;
  const int LHS_TILE_ROW_START = THIS_THREAD_ID / LHS_TILE_THREAD_PER_ROW;
  const int RHS_TILE_ROW_START = THIS_THREAD_ID / RHS_TILE_THREAD_PER_ROW;
  const int LHS_TILE_COL = THIS_THREAD_ID % LHS_TILE_THREAD_PER_ROW * 4;
  const int RHS_TILE_COL = THIS_THREAD_ID % RHS_TILE_THREAD_PER_ROW * 4;
  const int LHS_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / LHS_TILE_THREAD_PER_ROW;
  const int RHS_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / RHS_TILE_THREAD_PER_ROW;
  lhs = &lhs[(BLOCK_SIZE_M * BY) * k];
  rhs = &rhs[BLOCK_SIZE_N * BX];
  const int warpId = THIS_THREAD_ID / 32;
  const int laneId = THIS_THREAD_ID % 32;
  const int lhsTileIndex = warpId / 2 * 16 + laneId / 8 * 4;
  const int rhsTileIndex = warpId % 2 * 32 + laneId % 8 * 4;

// transfer lhs first tile block to shared memory
#pragma unroll
  for (int i = 0; i < BLOCK_SIZE_M; i += LHS_TILE_ROW_STRIDE) {
    int _idx = i / LHS_TILE_ROW_STRIDE * 4;
    FETCH_FLOAT4(lhsTransferRegister[_idx]) =
        FETCH_FLOAT4(lhs[OFFSET(LHS_TILE_ROW_START + i,  // row
                                LHS_TILE_COL,            // col
                                k)]);
    lhsShared[0][LHS_TILE_COL][LHS_TILE_ROW_START + i] = lhsTransferRegister[_idx];
    lhsShared[0][LHS_TILE_COL + 1][LHS_TILE_ROW_START + i] = lhsTransferRegister[_idx + 1];
    lhsShared[0][LHS_TILE_COL + 2][LHS_TILE_ROW_START + i] = lhsTransferRegister[_idx + 2];
    lhsShared[0][LHS_TILE_COL + 3][LHS_TILE_ROW_START + i] = lhsTransferRegister[_idx + 3];
  }

  //  transfer rhs first tile block to shared memory
#pragma unroll
  for (int i = 0; i < BLOCK_SIZE_K; i += RHS_TILE_ROW_STRIDE) {
    FETCH_FLOAT4(rhsShared[0][RHS_TILE_ROW_START + i][RHS_TILE_COL]) =
        FETCH_FLOAT4(rhs[OFFSET(RHS_TILE_ROW_START + i,  // row
                                RHS_TILE_COL,            // col
                                n)]);
  }
  __syncthreads();

  // load lhs from shared memory to register
  FETCH_FLOAT4(lhsRegister[0][0]) = FETCH_FLOAT4(lhsShared[0][0][lhsTileIndex]);
  FETCH_FLOAT4(lhsRegister[0][4]) = FETCH_FLOAT4(lhsShared[0][0][lhsTileIndex + 64]);

  // load rhs from shared memory to register
  FETCH_FLOAT4(rhsRegister[0][0]) = FETCH_FLOAT4(rhsShared[0][0][rhsTileIndex]);
  FETCH_FLOAT4(rhsRegister[0][4]) = FETCH_FLOAT4(rhsShared[0][0][rhsTileIndex + 64]);
}

#undef OFFSET
#undef FETCH_FLOAT4

}  // namespace kernel
}  // namespace rt
}  // namespace nncv