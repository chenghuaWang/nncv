#include "nncv/core/base.hpp"
#include "nncv/vm/cuda/vm_utils_cuda.cuh"

namespace nncv {
namespace vm {

extern "C" void PrintCudaDevicesInfo() {
  int device_cnt;
  hipGetDeviceCount(&device_cnt);
  if ((device_cnt == 0 && nncv::is_cuda_builtin) || (nncv::is_cuda_builtin == 0)) {
    printf(
        "Error: This version of nncv is not build for cuda or your machine has no cuda device\n");
    exit(EXIT_CUDA_ERROR);
  }
  printf("Info: found cuda device %d\n", device_cnt);
}

extern "C" void GetCudaDeviceProperty(CudaDeviceProperty_t& cdp, int idx) {
  hipDeviceProp_t device_prop;
  hipGetDeviceProperties(&device_prop, idx);
  // TODO
}

}  // namespace vm
}  // namespace nncv
