#include "nncv/vm/cuda/vm_alloc_cuda.cuh"

namespace nncv {
namespace vm {

extern "C" void _VmCudaMallocWarper(void** _devPtr, size_t _size) {
  NNCV_CUDA_CALL(hipMalloc(_devPtr, _size));
}

extern "C" void _VmCudaFreeWarper(void* _devPtr) { NNCV_CUDA_CALL(hipFree(_devPtr)); }

}  // namespace vm
}  // namespace nncv